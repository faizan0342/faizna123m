/*
   Copyright 2018 Lip Wee Yeo Amano

   Licensed under the Apache License, Version 2.0 (the "License");
   you may not use this file except in compliance with the License.
   You may obtain a copy of the License at

	   http://www.apache.org/licenses/LICENSE-2.0

   Unless required by applicable law or agreed to in writing, software
   distributed under the License is distributed on an "AS IS" BASIS,
   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
   See the License for the specific language governing permissions and
   limitations under the License.
*/

#include <hip/hip_runtime.h>
#include <iostream>
#include <cstring>
#include <string>

// Define this to turn on error checking
#define CUDA_ERROR_CHECK

#define CudaSafeCall(err)					__cudaSafeCall(err, __FILE__, __LINE__)
#define CudaCheckError(err, errMessage)		__cudaCheckError(err, errMessage)
#define CudaSyncAndCheckError(errMessage)	__cudaSyncAndCheckError(errMessage)

__host__ inline std::string __cudaSafeCall(hipError_t err, const char *file, const int line)
{
#ifdef CUDA_ERROR_CHECK
	if (hipSuccess != err)
		return hipGetErrorString(err);
	else
#endif //CUDA_ERROR_CHECK
		return "";
}

__host__ inline bool __cudaCheckError(hipError_t err, const char *errorMessage)
{
#ifdef CUDA_ERROR_CHECK
	if (err != hipSuccess)
	{
		auto errorMsgChar = hipGetErrorString(err);
		std::string errorMsg{ errorMsgChar };

		std::memcpy((void *)errorMessage, errorMsgChar, errorMsg.length());
		std::memset((void *)&errorMessage[errorMsg.length()], 0, 1);

		return false;
	}
#endif //CUDA_ERROR_CHECK

	return true;
}

__host__ inline bool __cudaSyncAndCheckError(const char *errorMessage)
{
	hipError_t response{ hipSuccess };
	std::string cudaErrors{ "" };

#ifdef CUDA_ERROR_CHECK
	response = hipGetLastError();
	if (response != hipSuccess)
	{
		while (response != hipSuccess)
		{
			if (!cudaErrors.empty()) cudaErrors += " <- ";
			cudaErrors += hipGetErrorString(response);
			response = hipGetLastError();
		}
		auto errorChar = cudaErrors.c_str();

		std::memcpy((void *)errorMessage, errorChar, cudaErrors.length());
		std::memset((void *)&errorMessage[cudaErrors.length()], 0, 1);

		return false;
	}
#endif //CUDA_ERROR_CHECK

	response = hipDeviceSynchronize();

	if (response != hipSuccess)
	{
		response = hipGetLastError();

		while (response != hipSuccess)
		{
			if (!cudaErrors.empty()) cudaErrors += " <- ";
			cudaErrors += hipGetErrorString(response);
			response = hipGetLastError();
		}
		auto errorChar = cudaErrors.c_str();

		std::memcpy((void *)errorMessage, errorChar, cudaErrors.length());
		std::memset((void *)&errorMessage[cudaErrors.length()], 0, 1);

		return false;
	}
	return true;
}
