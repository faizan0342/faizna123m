#include "hip/hip_runtime.h"
/*
   Copyright 2018 Lip Wee Yeo Amano

   Licensed under the Apache License, Version 2.0 (the "License");
   you may not use this file except in compliance with the License.
   You may obtain a copy of the License at

	   http://www.apache.org/licenses/LICENSE-2.0

   Unless required by applicable law or agreed to in writing, software
   distributed under the License is distributed on an "AS IS" BASIS,
   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
   See the License for the specific language governing permissions and
   limitations under the License.
*/

/*
* based off of https://github.com/Dunhili/SHA3-gpu-brute-force-cracker/blob/master/sha3.cu
*
* Author: Brian Bowden
* Date: 5/12/14
*
* This is the parallel version of SHA-3.
*/

#include "cudaErrorCheck.cu"
#include "cudaSolver.h"

typedef union
{
	uint2		uint2;
	uint64_t	uint64;
	uint8_t		uint8[UINT64_LENGTH];
} nonce_t;

__constant__ uint64_t d_midstate[25];
__constant__ uint64_t d_target[1];

__device__ __forceinline__ nonce_t bswap_64(nonce_t const input)
{
	nonce_t output;
	asm("{"
		"  prmt.b32 %0, %3, 0, 0x0123;"
		"  prmt.b32 %1, %2, 0, 0x0123;"
		"}" : "=r"(output.uint2.x), "=r"(output.uint2.y) : "r"(input.uint2.x), "r"(input.uint2.y));
	return output;
}

__device__ __forceinline__ nonce_t xor5(nonce_t const a, nonce_t const b, nonce_t const c, nonce_t const d, nonce_t const e)
{
	nonce_t output;
#if __CUDA_ARCH__ >= 500
	asm("{"
		"  lop3.b32 %0, %2, %4, %6, 0x96;"
		"  lop3.b32 %1, %3, %5, %7, 0x96;"
		"  lop3.b32 %0, %0, %8, %10, 0x96;"
		"  lop3.b32 %1, %1, %9, %11, 0x96;"
		"}" : "=r"(output.uint2.x), "=r"(output.uint2.y)
		: "r"(a.uint2.x), "r"(a.uint2.y), "r"(b.uint2.x), "r"(b.uint2.y), "r"(c.uint2.x), "r"(c.uint2.y), "r"(d.uint2.x), "r"(d.uint2.y), "r"(e.uint2.x), "r"(e.uint2.y));
#else
	asm("{"
		"  xor.b64 %0, %1, %2;"
		"  xor.b64 %0, %0, %3;"
		"  xor.b64 %0, %0, %4;"
		"  xor.b64 %0, %0, %5;"
		"}" : "=l"(output.uint64) : "l"(a.uint64), "l"(b.uint64), "l"(c.uint64), "l"(d.uint64), "l"(e.uint64));
#endif
	return output;
}

__device__ __forceinline__ nonce_t xor3(nonce_t const a, nonce_t const b, nonce_t const c)
{
	nonce_t output;
#if __CUDA_ARCH__ >= 500
	asm("{"
		"  lop3.b32 %0, %2, %4, %6, 0x96;"
		"  lop3.b32 %1, %3, %5, %7, 0x96;"
		"}" : "=r"(output.uint2.x), "=r"(output.uint2.y)
		: "r"(a.uint2.x), "r"(a.uint2.y), "r"(b.uint2.x), "r"(b.uint2.y), "r"(c.uint2.x), "r"(c.uint2.y));
#else
	asm("{"
		"  xor.b64 %0, %1, %2;"
		"  xor.b64 %0, %0, %3;"
		"}" : "=l"(output.uint64) : "l"(a.uint64), "l"(b.uint64), "l"(c.uint64));
#endif
	return output;
}

__device__ __forceinline__ nonce_t chi(nonce_t const a, nonce_t const b, nonce_t const c)
{
	nonce_t output;
#if __CUDA_ARCH__ >= 500
	asm("{"
		"  lop3.b32 %0, %2, %4, %6, 0xD2;"
		"  lop3.b32 %1, %3, %5, %7, 0xD2;"
		"}" : "=r"(output.uint2.x), "=r"(output.uint2.y)
		: "r"(a.uint2.x), "r"(a.uint2.y), "r"(b.uint2.x), "r"(b.uint2.y), "r"(c.uint2.x), "r"(c.uint2.y));
#else
	output.uint64 = a.uint64 ^ ((~b.uint64) & c.uint64);
#endif
	return output;
}

__device__ __forceinline__ nonce_t rotl(nonce_t input, uint32_t const offset)
{
#if __CUDA_ARCH__ >= 320
	asm("{"
		"  .reg .b32 tmp;"
		"  shf.l.wrap.b32 tmp, %1, %0, %2;"
		"  shf.l.wrap.b32 %1, %0, %1, %2;"
		"  mov.b32 %0, tmp;"
		"}" : "+r"(input.uint2.x), "+r"(input.uint2.y) : "r"(offset));
#else
	input.uint64 = (input.uint64 << offset) ^ (input.uint64 >> (64u - offset));
#endif
	return input;
}

__device__ __forceinline__ nonce_t rotr(nonce_t input, uint32_t const offset)
{
#if __CUDA_ARCH__ >= 320
	asm("{"
		"  .reg .b32 tmp;"
		"  shf.r.wrap.b32 tmp, %0, %1, %2;"
		"  shf.r.wrap.b32 %1, %1, %0, %2;"
		"  mov.b32 %0, tmp;"
		"}" : "+r"(input.uint2.x), "+r"(input.uint2.y) : "r"(offset));
#else
	input.uint64 = (input.uint64 >> offset) ^ (input.uint64 << (64u - offset));
#endif
	return input;
}

__global__ void hashMidstate(uint64_t *__restrict__ solutions, uint32_t *__restrict__ solutionCount, uint32_t maxSolutionCount, uint64_t startPosition)
{
	nonce_t nonce, state[25], C[5], D[5], n[11];
	nonce.uint64 = blockDim.x * blockIdx.x + threadIdx.x + startPosition;

	n[0] = rotl(nonce, 7);
	n[1] = rotl(n[0], 1);
	n[2] = rotl(n[1], 6);
	n[3] = rotl(n[2], 2);
	n[4] = rotl(n[3], 4);
	n[5] = rotl(n[4], 7);
	n[6] = rotl(n[5], 12);
	n[7] = rotl(n[6], 5);
	n[8] = rotl(n[7], 11);
	n[9] = rotl(n[8], 7);
	n[10] = rotl(n[9], 1);

	C[0].uint64 = d_midstate[0];
	C[1].uint64 = d_midstate[1];
	C[2].uint64 = d_midstate[2] ^ n[7].uint64;
	C[3].uint64 = d_midstate[3];
	C[4].uint64 = d_midstate[4] ^ n[2].uint64;
	state[0].uint64 = chi(C[0], C[1], C[2]).uint64 ^ Keccak_f1600_RC[0];
	state[1] = chi(C[1], C[2], C[3]);
	state[2] = chi(C[2], C[3], C[4]);
	state[3] = chi(C[3], C[4], C[0]);
	state[4] = chi(C[4], C[0], C[1]);

	C[0].uint64 = d_midstate[5];
	C[1].uint64 = d_midstate[6] ^ n[4].uint64;
	C[2].uint64 = d_midstate[7];
	C[3].uint64 = d_midstate[8];
	C[4].uint64 = d_midstate[9] ^ n[9].uint64;
	state[5] = chi(C[0], C[1], C[2]);
	state[6] = chi(C[1], C[2], C[3]);
	state[7] = chi(C[2], C[3], C[4]);
	state[8] = chi(C[3], C[4], C[0]);
	state[9] = chi(C[4], C[0], C[1]);

	C[0].uint64 = d_midstate[10];
	C[1].uint64 = d_midstate[11] ^ n[0].uint64;
	C[2].uint64 = d_midstate[12];
	C[3].uint64 = d_midstate[13] ^ n[1].uint64;
	C[4].uint64 = d_midstate[14];
	state[10] = chi(C[0], C[1], C[2]);
	state[11] = chi(C[1], C[2], C[3]);
	state[12] = chi(C[2], C[3], C[4]);
	state[13] = chi(C[3], C[4], C[0]);
	state[14] = chi(C[4], C[0], C[1]);

	C[0].uint64 = d_midstate[15] ^ n[5].uint64;
	C[1].uint64 = d_midstate[16];
	C[2].uint64 = d_midstate[17];
	C[3].uint64 = d_midstate[18] ^ n[3].uint64;
	C[4].uint64 = d_midstate[19];
	state[15] = chi(C[0], C[1], C[2]);
	state[16] = chi(C[1], C[2], C[3]);
	state[17] = chi(C[2], C[3], C[4]);
	state[18] = chi(C[3], C[4], C[0]);
	state[19] = chi(C[4], C[0], C[1]);

	C[0].uint64 = d_midstate[20] ^ n[10].uint64;
	C[1].uint64 = d_midstate[21] ^ n[8].uint64;
	C[2].uint64 = d_midstate[22] ^ n[6].uint64;
	C[3].uint64 = d_midstate[23];
	C[4].uint64 = d_midstate[24];
	state[20] = chi(C[0], C[1], C[2]);
	state[21] = chi(C[1], C[2], C[3]);
	state[22] = chi(C[2], C[3], C[4]);
	state[23] = chi(C[3], C[4], C[0]);
	state[24] = chi(C[4], C[0], C[1]);

#if __CUDA_ARCH__ >= 350
#	pragma unroll
#endif
	for (int i{ 1 }; i < 23; ++i)
	{
		C[1] = xor5(state[0], state[5], state[10], state[15], state[20]);
		C[2] = xor5(state[1], state[6], state[11], state[16], state[21]);
		C[3] = xor5(state[2], state[7], state[12], state[17], state[22]);
		C[4] = xor5(state[3], state[8], state[13], state[18], state[23]);
		C[0] = xor5(state[4], state[9], state[14], state[19], state[24]);

#if __CUDA_ARCH__ >= 350
		D[0] = rotl(C[2], 1);
		state[0] = xor3(state[0], D[0], C[0]);
		state[5] = xor3(state[5], D[0], C[0]);
		state[10] = xor3(state[10], D[0], C[0]);
		state[15] = xor3(state[15], D[0], C[0]);
		state[20] = xor3(state[20], D[0], C[0]);

		D[1] = rotl(C[3], 1);
		state[1] = xor3(state[1], D[1], C[1]);
		state[6] = xor3(state[6], D[1], C[1]);
		state[11] = xor3(state[11], D[1], C[1]);
		state[16] = xor3(state[16], D[1], C[1]);
		state[21] = xor3(state[21], D[1], C[1]);

		D[2] = rotl(C[4], 1);
		state[2] = xor3(state[2], D[2], C[2]);
		state[7] = xor3(state[7], D[2], C[2]);
		state[12] = xor3(state[12], D[2], C[2]);
		state[17] = xor3(state[17], D[2], C[2]);
		state[22] = xor3(state[22], D[2], C[2]);

		D[3] = rotl(C[0], 1);
		state[3] = xor3(state[3], D[3], C[3]);
		state[8] = xor3(state[8], D[3], C[3]);
		state[13] = xor3(state[13], D[3], C[3]);
		state[18] = xor3(state[18], D[3], C[3]);
		state[23] = xor3(state[23], D[3], C[3]);

		D[4] = rotl(C[1], 1);
		state[4] = xor3(state[4], D[4], C[4]);
		state[9] = xor3(state[9], D[4], C[4]);
		state[14] = xor3(state[14], D[4], C[4]);
		state[19] = xor3(state[19], D[4], C[4]);
		state[24] = xor3(state[24], D[4], C[4]);
#else
		for (int x{ 0 }; x < 5; ++x)
		{
			D[x].uint64 = rotl(C[(x + 2) % 5], 1).uint64 ^ C[x].uint64;
			state[x].uint64 = state[x].uint64 ^ D[x].uint64;
			state[x + 5].uint64 = state[x + 5].uint64 ^ D[x].uint64;
			state[x + 10].uint64 = state[x + 10].uint64 ^ D[x].uint64;
			state[x + 15].uint64 = state[x + 15].uint64 ^ D[x].uint64;
			state[x + 20].uint64 = state[x + 20].uint64 ^ D[x].uint64;
		}
#endif

		C[0] = state[1];
		state[1] = rotr(state[6], 20);
		state[6] = rotl(state[9], 20);
		state[9] = rotr(state[22], 3);
		state[22] = rotr(state[14], 25);
		state[14] = rotl(state[20], 18);
		state[20] = rotr(state[2], 2);
		state[2] = rotr(state[12], 21);
		state[12] = rotl(state[13], 25);
		state[13] = rotl(state[19], 8);
		state[19] = rotr(state[23], 8);
		state[23] = rotr(state[15], 23);
		state[15] = rotl(state[4], 27);
		state[4] = rotl(state[24], 14);
		state[24] = rotl(state[21], 2);
		state[21] = rotr(state[8], 9);
		state[8] = rotr(state[16], 19);
		state[16] = rotr(state[5], 28);
		state[5] = rotl(state[3], 28);
		state[3] = rotl(state[18], 21);
		state[18] = rotl(state[17], 15);
		state[17] = rotl(state[11], 10);
		state[11] = rotl(state[7], 6);
		state[7] = rotl(state[10], 3);
		state[10] = rotl(C[0], 1);

#if __CUDA_ARCH__ >= 350
#	pragma unroll
#endif
		for (int x{ 0 }; x < 25; x += 5)
		{
			C[0] = state[x];
			C[1] = state[x + 1];
			C[2] = state[x + 2];
			C[3] = state[x + 3];
			C[4] = state[x + 4];
			state[x] = chi(C[0], C[1], C[2]);
			state[x + 1] = chi(C[1], C[2], C[3]);
			state[x + 2] = chi(C[2], C[3], C[4]);
			state[x + 3] = chi(C[3], C[4], C[0]);
			state[x + 4] = chi(C[4], C[0], C[1]);
		}

		state[0].uint64 = state[0].uint64 ^ Keccak_f1600_RC[i];
	}

	C[1] = xor5(state[0], state[5], state[10], state[15], state[20]);
	C[2] = xor5(state[1], state[6], state[11], state[16], state[21]);
	C[3] = xor5(state[2], state[7], state[12], state[17], state[22]);
	C[4] = xor5(state[3], state[8], state[13], state[18], state[23]);
	C[0] = xor5(state[4], state[9], state[14], state[19], state[24]);

	D[0] = rotl(C[2], 1);
	D[1] = rotl(C[3], 1);
	D[2] = rotl(C[4], 1);

	state[0] = xor3(state[0], D[0], C[0]);
	state[6] = xor3(state[6], D[1], C[1]);
	state[12] = xor3(state[12], D[2], C[2]);
	state[6] = rotr(state[6], 20);
	state[12] = rotr(state[12], 21);

	state[0].uint64 = chi(state[0], state[6], state[12]).uint64 ^ Keccak_f1600_RC[23];

	if (bswap_64(state[0]).uint64 <= d_target[0]) // LTE is allowed because d_target is high 64 bits of uint256 (let CPU do the verification)
	{
		if (*solutionCount < maxSolutionCount)
		{
			solutions[*solutionCount] = nonce.uint64;
			(*solutionCount)++;
		}
	}
}

// --------------------------------------------------------------------
// CudaSolver
// --------------------------------------------------------------------

namespace CUDASolver
{
	void CudaSolver::PushHigh64Target(uint64_t *high64Target, const char *errorMessage)
	{
		CudaCheckError(hipMemcpyToSymbol(HIP_SYMBOL(d_target), high64Target, UINT64_LENGTH, 0, hipMemcpyHostToDevice), errorMessage);
	}

	void CudaSolver::PushMidState(sponge_ut *midState, const char *errorMessage)
	{
		CudaCheckError(hipMemcpyToSymbol(HIP_SYMBOL(d_midstate), midState, SPONGE_LENGTH, 0, hipMemcpyHostToDevice), errorMessage);
	}

	void CudaSolver::HashMidState(DeviceCUDA *device, const char *errorMessage)
	{
		hashMidstate<<<device->Grid, device->Block>>>(device->SolutionsDevice, device->SolutionCountDevice, device->MaxSolutionCount, device->WorkPosition);
		CudaSyncAndCheckError(errorMessage);
	}
}